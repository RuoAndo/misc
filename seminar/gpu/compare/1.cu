#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 256

__global__ void matrix_vector_multi_gpu_1_1(float *A_d, float *B_d, float *C_d)
{
	int i,j;

	for(j=0;j<N;j++){
		A_d[j] = 0.0F;
		for(i=0;i<N;i++) {
		 A_d[j] = A_d[j]+B_d[j*N+i]*C_d[i];
		 }
	}
}

__global__ void matrix_vector_multi_gpu_1_1_sh(float *A_d, float *B_d, float *C_d)
{
	int i;

	__shared__ float tmp_c[N];

	tmp_c[threadIdx.x] = C_d[threadIdx.x];
	__syncthreads();

	A_d[threadIdx.x] = 0.0F;

	for(i=0;i<N;i++){
		A_d[threadIdx.x] = A_d[threadIdx.x] + B_d[threadIdx.x*N+1]*tmp_c[i];
	}
}

int main(int argc, char **argv)
{
    // set up device
    int dev = 0;

    unsigned int t, travdirtime; 

    int i,j;
    float A[N], B[N*N], C[N];
    float *A_d, *B_d, *C_d;

    dim3 blocks(1,1,1);
    dim3 threads(1,1,1);

    for(j=0;j<N;j++) {
        for(i=0;i<N;i++) {
	   B[j*N+i] = ((float)j)/256.0;
	   }
    }	   

    for(j=0;j<N;j++)
	C[j] = 1.0F;
	
    hipMalloc((void **)&A_d, N*sizeof(float));
    hipMalloc((void **)&B_d, N*N*sizeof(float));
    hipMalloc((void **)&C_d, N*sizeof(float));

    hipMemcpy(A_d, A, N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(B_d, B, N*N*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(C_d, C, N*sizeof(float), hipMemcpyHostToDevice);

    matrix_vector_multi_gpu_1_1<<<blocks, threads>>>(A_d, B_d, C_d);
    matrix_vector_multi_gpu_1_1_sh<<<blocks, threads>>>(A_d, B_d, C_d);

    hipFree(A_d);
    hipFree(B_d);
    hipFree(C_d);

    return EXIT_SUCCESS;
}
